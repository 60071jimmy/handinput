#include "hip/hip_runtime.h"
#define _SIZE_T_DEFINED 
#ifndef __HIPCC__ 
#define __HIPCC__ 
#endif 
#ifndef __cplusplus 
#define __cplusplus 
#endif

#include "stdafx.h"

__global__ void SkinKernel(const uchar *src, uchar *dst, const int rows, const int cols,
                           const int srcStep, const int srcChannels, const int dstStep) {
  int r = blockIdx.y * blockDim.y + threadIdx.y;
  int c = blockIdx.x * blockDim.x + threadIdx.x;
  if (r >= rows || c >= cols) return;

  int iSrc = r * srcStep + c * srcChannels;
  int iDst = r * dstStep + c;

  int y = src[iSrc];
  int cr = src[iSrc + 1];
  int cb = src[iSrc + 2];

  cb -= 109;
  cr -= 152;
  int x1 = (819 * cr - 614 * cb) / 32 + 51;
  int y1 = (819 * cr + 614 * cb) / 32 + 77;
  x1 = x1 * 41 / 1024;
  y1 = y1 * 73 / 1024;
  int value = x1 * x1 + y1 * y1;
  if (y < 100)
    dst[iDst] = (value < 700) ? (unsigned char)255 : (unsigned char)0;
  else
    dst[iDst] = (value < 850) ? (unsigned char)255 : (unsigned char)0; 

}

extern "C" {

__declspec(dllexport) void __stdcall FilterSkin(void* src_, void* dst_) {
  using cv::gpu::GpuMat;

  GpuMat* src = reinterpret_cast<GpuMat*>(src_);
  GpuMat* dst = reinterpret_cast<GpuMat*>(dst_);

  dim3 dimBlock(20, 15);
  int rows = src->rows;
  int cols = src->cols;
  int srcStep = src->step1();
  int dstStep = dst->step1();
  int srcChannels = src->channels();
  
  // Find ceilings.
  dim3 dimGrid((cols + dimBlock.x - 1) / dimBlock.x, (rows + dimBlock.y - 1) / dimBlock.y);
  SkinKernel<<<dimGrid, dimBlock>>>(src->ptr(), dst->ptr(), rows, cols, srcStep, srcChannels,
                                    dstStep);
  hipDeviceSynchronize();
}

}


